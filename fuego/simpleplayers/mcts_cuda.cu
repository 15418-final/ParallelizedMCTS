#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cstdio>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include "mcts.h"
#include "CudaGo.h"
#include "deque.h"

template <typename T>
struct KernelArray
{
	T* _array;
	int _size;
};

//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

__constant__ int MAX_TRIAL = 5;
__constant__ int THREAD_NUM = 32;
int MAX_TRIAL_H = 5;


__device__ bool checkAbortCuda(double timeLeft, clock_t start);
__device__ bool checkAbort();
__device__ Deque<Point*>* generateAllMoves(CudaBoard* cur_board);
__device__ void deleteAllMoves(Deque<Point*>* moves);
__global__ void run_simulation(KernelArray<Point> seq, int* win_increase, int bd_size);
__device__ CudaBoard* get_board(KernelArray<Point> seq, int bd_size);

void getMemoryInfo();

template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec);

SgPoint Mcts::run() {
	mcts_timer.Start();
	while (true) {
		run_iteration(root);
		if (checkAbort()) break;
	}
	double maxv = 0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}
	if (best == NULL) {
		return SG_NULLMOVE;
	}
	// std::cout << "Total simulation runs:" << totalSimu << std::endl;
	return best->get_sequence().back().ToSgPoint();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;
	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation

__global__ void run_simulation(int* iarray, int* jarray, int len, int* win_increase, int bd_size) {
	CudaBoard* board = new CudaBoard(bd_size);
	for (int i = 0; i < len; i++) {
		Point* p = new Point(iarray[i], jarray[i]);
		board->update_board(p);
		delete p;
	}
	COLOR cur_player = board->ToPlay();
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	(*win_increase) = 0;

	for (int i = 0; i < MAX_TRIAL; i++) {
		CudaBoard* cur_board = new CudaBoard(*board);

		if (index == 0)
			printf("Start a simulation\n");
		int time = 0;
		while (true) {
			Deque<Point*>* moves_vec = generateAllMoves(cur_board);
			// printf("generate moves done:%d\n",moves_vec->size());
			if (cur_board->EndOfGame() || moves_vec->size() == 0) {
				printf("game end normally\n");
				break;
			}
			//why nxt_move length can be zero? what does endofgame do above?
			Point* nxt_move = moves_vec->front();
			cur_board->update_board(nxt_move);
			deleteAllMoves(moves_vec);
			delete moves_vec;
			time++;
			printf("time in while:%d\n",time);
		}
		
		int score = cur_board->score(); // Komi set to 0
		if ((score > 0 && cur_player == BLACK)
		        || (score < 0 && cur_player == WHITE)) {
			(*win_increase)++;

		}
		// totalSimu ++;
		
		if (index == 0)
			printf("%d win\n", score > 0 ? 1: 2);
		//printf("run simulation done\n");
		delete cur_board;
	}
	if (index == 0)
		printf("run_simulation done\n");
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	// std::cout<< "node->sims:"<< node->sims <<std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(),bd_size);

	std::vector<Point*> moves_vec = generateAllMoves(cur_board);
	// std::cout<<"moves generated:"<< moves_vec.size() <<std::endl;
	while (moves_vec.size() > 0) {
		Point* nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), *nxt_move));
		moves_vec.pop_back();
		delete nxt_move;
	}
	std::cout<<"children add done"<<std::endl;
	deleteAllMoves(moves_vec);
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			//	std::cout<<"select f:"<<f<<std::endl;
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);
			std::cout<<"expand f end:"<<f<<std::endl;

			std::vector<TreeNode*> children = f->get_children();
			for (size_t i = 0; i < children.size(); i++) {

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				int* cuda_win_increase = NULL;
				hipMalloc((void **)&cuda_win_increase, sizeof(int));

				// std::cout<<"Cuda malloc done"<<std::endl;

				std::vector<Point> sequence = children[i]->get_sequence();
				int len = sequence.size();
				int* c_i = new int[len];
				int* c_j = new int[len];
				int* c_i_d; // device
				int* c_j_d; // device
				for (int it = 0; it < len; it++) {
					c_i[it] = sequence[it].i;
					c_j[it] = sequence[it].j;
				}

				hipMalloc(&c_i_d, sizeof(int)*len);
    			hipMalloc(&c_j_d, sizeof(int)*len);
    			hipMemcpy(c_i_d, c_i, sizeof(int)*len, hipMemcpyHostToDevice); 
    			hipMemcpy(c_j_d, c_j, sizeof(int)*len, hipMemcpyHostToDevice); 
				

				CudaBoard* board = get_board(sequence, bd_size);
				// board->print_board();

				std::cout<<"ready to run cuda code run_simulation()"<<std::endl;
				hipEventRecord(start);
				run_simulation<<<1,1>>>(c_i_d, c_j_d, len, cuda_win_increase, bd_size);
				hipEventRecord(stop);
				printf("return : %s\n",hipGetErrorString(hipDeviceSynchronize()));
				//hipFree(cudaDeviceNode);

				getMemoryInfo();
				hipDeviceSynchronize();
				int* win_increase = new int[1];
				hipMemcpy(win_increase, cuda_win_increase, sizeof(int), hipMemcpyDeviceToHost);
				hipEventSynchronize(stop);
				float milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start, stop);

				printf("time: %f\n", milliseconds);
				printf("win: %d\n", *win_increase);

				hipFree(c_i_d);
				hipFree(c_j_d);

				children[i]->wins += *win_increase;
				children[i]->sims += MAX_TRIAL_H;
				//printf("win:%d, sims:%d\n", children[i]->wins, children[i]->sims);
				back_propagation(children[i], *win_increase, MAX_TRIAL_H);
				delete win_increase;
				if(checkAbort())break;
			}
		}
		if (checkAbort()) break;
	}

	std::cout << "run_iteration end:" << std::endl;
}

bool Mcts::checkAbort() {
	if (!abort) {
		abort = mcts_timer.GetTime() > maxTime;
	}
	return abort;
}

__device__ bool checkAbortCuda(double timeLeft, clock_t start){
	if(timeLeft < clock64() - start){
		return true;
	}
	return false;
}

__device__ Deque<Point*>* generateAllMoves(CudaBoard* cur_board) {
	Deque<Point*>* moves_vec = cur_board->get_next_moves_device();
	int len = moves_vec->size();

	// TODO : rand in cuda
	// if (len != 0) {
	// 	srand (time(NULL));
	// 	int swapIndex = rand() % len;
	// 	Point* temp = (*moves_vec)[moves_vec->begin()];
	// 	(*moves_vec)[moves_vec->begin()] = (*moves_vec)[moves_vec->begin() + swapIndex];
	// 	(*moves_vec)[moves_vec->begin() + swapIndex] = temp;
	// }

	return moves_vec;
}

std::vector<Point*> Mcts::generateAllMoves(CudaBoard* cur_board) {
	std::vector<Point*> moves_vec = cur_board->get_next_moves_host();
	int len = moves_vec.size();

	// if (len != 0) {
	// 	srand (time(NULL));
	// 	int swapIndex = rand() % len;
	// 	Point* temp = moves_vec[0];
	// 	moves_vec[0] = moves_vec[swapIndex];
	// 	moves_vec[swapIndex] = temp;
	// }
	return moves_vec;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board(&(*it));
	}
	return bd;
}

__device__ CudaBoard* get_board(KernelArray<Point> sequence, int bd_size) {
	CudaBoard *bd = new CudaBoard(bd_size);
	for (int i = 0; i < sequence._size; i++) {
		bd->update_board(&sequence._array[i]);
	}

	return bd;
}

__device__ void deleteAllMoves(Deque<Point*>* moves) {
	Deque<Point*>::iterator it = moves->begin();
	for (; it != moves->end(); it++) {
		Point* p = *it;
		delete p;
	} 
}

void getMemoryInfo(){
	size_t free_byte ;

	size_t total_byte ;

	hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

	if ( hipSuccess != cuda_status ) {

		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );

		exit(1);

	}
}

void Mcts::deleteAllMoves(std::vector<Point*> moves) {
	for (std::vector<Point*>::iterator it = moves.begin(); it != moves.end(); it++) {
		delete *it;
	}
}

template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec)
{
    KernelArray<T> kArray;
    kArray._array = thrust::raw_pointer_cast(&dVec[0]);
    kArray._size  = (int) dVec.size();
 
    return kArray;
}

