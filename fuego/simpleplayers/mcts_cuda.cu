#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cstdio>
// #include <ctime>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include "mcts.h"
#include "CudaGo.h"
#include "deque.h"

template <typename T>
struct KernelArray
{
	T* _array;
	int _size;
};

//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;
__constant__ int MAX_TRIAL = 500;
__constant__ int THREAD_NUM = 32;

int MAX_TRIAL_H = 500;


__device__ bool checkAbort();
__device__ Deque<Point*>* generateAllMoves(CudaBoard* cur_board);
__device__ void deleteAllMoves(Deque<Point*>* moves);
__global__ void run_simulation(KernelArray<Point>, int* win_increase, int bd_size);
__device__ CudaBoard* get_board(KernelArray<Point> seq, int bd_size);

template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec);

SgPoint Mcts::run() {
	// mcts_timer.Start();
	while (true) {
		run_iteration(root);
		if (checkAbort()) break;
	}
	double maxv = 0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}
	if (best == NULL) {
		return SG_NULLMOVE;
	}
	// std::cout << "Total simulation runs:" << totalSimu << std::endl;
	return best->get_sequence().back().ToSgPoint();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;
	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation
__global__ void run_simulation(KernelArray<Point> seq, int* win_increase, int bd_size) {
	CudaBoard* board = get_board(seq, bd_size);
	COLOR cur_player = board->ToPlay();

	
	int wins = 0;
	for (int i = 0; i < MAX_TRIAL; i++) {
		// bool timeout = false;
		CudaBoard* cur_board = new CudaBoard(*board);
		clock_t start = clock();
		while (true) {
			Deque<Point*>* moves_vec = generateAllMoves(cur_board);
			if (cur_board->EndOfGame() || moves_vec->size() == 0) {
				break;
			}
			//why nxt_move length can be zero? what does endofgame do above?
			// std::cout << "moves_vec length:" << moves_vec->Length() << std::endl;
			Point* nxt_move = (*moves_vec)[moves_vec->begin()];
			
			cur_board->update_board(nxt_move);
			deleteAllMoves(moves_vec);
			// if (checkAbort()) {
			// 	timeout = true;
			// 	break;
			// }
		}
		if (true) {
			int score = cur_board->score(); // Komi set to 0
			if ((score > 0 && cur_player == BLACK)
			        || (score < 0 && cur_player == WHITE)) {
				wins++;
			}
			// totalSimu ++;
		}
		delete cur_board;
	}

	return;
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(),bd_size);

	std::vector<Point*> moves_vec = generateAllMoves(cur_board);
	while (moves_vec.size() > 0) {
		Point* nxt_move = moves_vec.front();
		node->add_children(new TreeNode(node->get_sequence(), *nxt_move));
	}

	deleteAllMoves(moves_vec);
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			//	std::cout<<"select f:"<<f<<std::endl;
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);
			//std::cout<<"expand f end:"<<f<<std::endl;

			std::vector<TreeNode*> children = f->get_children();
			for (size_t i = 0; i < children.size(); i++) {
				// TreeNode* cudaDeviceNode = NULL;
				// int* cuda_win_increase = NULL;
				// // Use cuda to parallelize
				// hipMalloc((void **)&cudaDeviceNode, sizeof(*children[i]));
				// hipMalloc((void **)&cuda_win_increase, sizeof(int));
				// hipMemcpy(cudaDeviceNode, children[i], sizeof(*children[i]), hipMemcpyHostToDevice);

				int* cuda_win_increase = NULL;
				hipMalloc((void **)&cuda_win_increase, sizeof(int));

				thrust::device_vector<Point> dec_seq(children[i]->get_sequence());
				
				run_simulation<<<1,1>>>(convertToKernel(dec_seq), cuda_win_increase, bd_size);
				//hipFree(cudaDeviceNode);

				int* win_increase = new int[1];
				hipMemcpy(win_increase, cuda_win_increase, sizeof(int), hipMemcpyDeviceToHost);

				children[i]->wins += *win_increase;
				children[i]->sims += MAX_TRIAL_H;
				back_propagation(children[i], *win_increase, MAX_TRIAL_H);
				delete cuda_win_increase;
				delete win_increase;
				if(checkAbort())break;
			}
		}
		if (checkAbort()) break;
	}

	std::cout << "run_iteration end:" << std::endl;
}

bool Mcts::checkAbort() {
	if (!abort) {
		// abort = mcts_timer.GetTime() > maxTime;
	}
	return abort;
}

__device__ Deque<Point*>* generateAllMoves(CudaBoard* cur_board) {
	Deque<Point*>* moves_vec = cur_board->get_next_moves_device();
	int len = moves_vec->size();

	// TODO : rand in cuda
	// if (len != 0) {
	// 	srand (time(NULL));
	// 	int swapIndex = rand() % len;
	// 	Point* temp = (*moves_vec)[moves_vec->begin()];
	// 	(*moves_vec)[moves_vec->begin()] = (*moves_vec)[moves_vec->begin() + swapIndex];
	// 	(*moves_vec)[moves_vec->begin() + swapIndex] = temp;
	// }

	return moves_vec;
}

std::vector<Point*> Mcts::generateAllMoves(CudaBoard* cur_board) {
	std::vector<Point*> moves_vec = cur_board->get_next_moves_host();
	int len = moves_vec.size();

	// if (len != 0) {
	// 	srand (time(NULL));
	// 	int swapIndex = rand() % len;
	// 	Point* temp = moves_vec[0];
	// 	moves_vec[0] = moves_vec[swapIndex];
	// 	moves_vec[swapIndex] = temp;
	// }
	return moves_vec;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board(&(*it));
	}
	return bd;
}

__device__ CudaBoard* get_board(KernelArray<Point> sequence, int bd_size) {
	CudaBoard *bd = new CudaBoard(bd_size);
	for (int i = 0; i < sequence._size; i++) {
		bd->update_board(&sequence._array[i]);
	}

	return bd;
}

__device__ void deleteAllMoves(Deque<Point*>* moves) {
	for (int i = moves->begin(); i <= moves->end(); i++) {
		Point* p = (*moves)[i];
		delete p;
	} 
}

void Mcts::deleteAllMoves(std::vector<Point*> moves) {
	for (std::vector<Point*>::iterator it = moves.begin(); it != moves.end(); it++) {
		delete *it;
	}
}

template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec)
{
    KernelArray<T> kArray;
    kArray._array = thrust::raw_pointer_cast(&dVec[0]);
    kArray._size  = (int) dVec.size();
 
    return kArray;
}

