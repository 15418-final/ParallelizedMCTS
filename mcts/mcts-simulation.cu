#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


void transfer_board(GoBoard& input, GoBoard& output){
	GoBoard* device_input;
	GoBoard* device_output;
	hipMalloc((void **)&device_output, sizeof(*input));
	hipMalloc((void **)&device_input, sizeof(*input));
	hipMemcpy(device_input, &input, sizeof(*input), 
               hipMemcpyHostToDevice);
	
}

__global__ void run_simulation(TreeNode* node){

}
