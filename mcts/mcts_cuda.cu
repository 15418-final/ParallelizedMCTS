#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include "mcts.h"
#include "CudaGo.h"
#include "deque.h"
#include "point.h"

//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

int MAX_TRIAL_H = 50;


__device__ bool checkAbort();
__global__ void run_simulation(int* iarray, int* jarray, int len, int* win_increase, Point* parray, int bd_size, unsigned int seed);
__device__ __host__ Point* createPoints(int bd_size);
__device__ __host__ void deletePoints(Point*** point, int bd_size);
__device__ void deleteAllMoves(Deque<Point*>* moves);

void memoryUsage();

Point Mcts::run() {
	// mcts_timer.Start();
	size_t heapszie = 32 * 1024 * 1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, heapszie);

	clock_t start = clock();
	while (true) {
		run_iteration(root);
		//	if (checkAbort()) break;
		clock_t end = clock();
		if ( ((end - start) / (double)(CLOCKS_PER_SEC / 1000)) > 10 * 1000) break;
	}
	double maxv = -1.0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}
	// std::cout << "Total simulation runs:" << totalSimu << std::endl;
	return best->get_sequence().back();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;

	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation
__global__ void run_simulation(int* iarray, int* jarray, int len, int* win_increase, Point* parray, int bd_size, unsigned int seed) {
	// TODO: use shared memory for point
	// __shared__ Point* point;
	
	// if (threadIdx.x == 0) {
	// 	memcpy(point, parray, sizeof(Point)*(bd_size+2)*(bd_size+2));
	// }
	// __syncthreads();

	Point* point = createPoints(bd_size);
	CudaBoard* board = new CudaBoard(bd_size);
	for (int i = 0; i < len; i++) {
		board->update_board(point[iarray[i]*(bd_size+2)+ jarray[i]], point);
	}

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	COLOR player = board->ToPlay();
	hiprandState_t state;
	hiprand_init(seed + index, 0, 0, &state);

	// bool timeout = false;
	*win_increase = 0;
	int step = 0;
	while (true && step < 300) {
		Deque<Point>* moves = board->get_next_moves_device(point);
		if (moves->size() == 0) {
			break;
		}

		Point nxt_move = (*moves)[hiprand(&state) % moves->size()];
		//Point* nxt_move = moves->front();
		board->update_board(nxt_move, point);
		step++;
	}

	int score = board->score(); // Komi set to 0
	if ((score > 0 && player == BLACK)
	        || (score < 0 && player == WHITE)) {
		(*win_increase)++;
	}

	//printf("id:%d, step:%d\n", index, step);
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(), bd_size);

	std::vector<Point> moves_vec = generateAllMoves(cur_board);
	std::cout << "moves generated:" << moves_vec.size() << std::endl;
	while (moves_vec.size() > 0) {
		Point nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), nxt_move));
		moves_vec.pop_back();
	}
	std::cout << "children add done" << std::endl;
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	int total = bd_size * bd_size;
	int* c_i = new int[total];
	int* c_j = new int[total];
	int* c_i_d; // device
	int* c_j_d; // device

	Point* points = createPoints(bd_size);

	std::cout << "run_iteration start:" << std::endl;

	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			//	std::cout<<"select f:"<<f<<std::endl;
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);

			std::vector<TreeNode*> children = f->get_children();
			for (size_t i = 0; i < children.size(); i++) {
				std::vector<Point> sequence = children[i]->get_sequence();
				int len = sequence.size();

				for (int it = 0; it < len; it++) {
					c_i[it] = sequence[it].i;
					c_j[it] = sequence[it].j;
				}

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				int* cuda_win_increase = NULL;
				Point* cuda_points = NULL;

				hipMalloc(&cuda_win_increase, sizeof(int));
				hipMalloc(&c_i_d, sizeof(int)*len);
				hipMalloc(&c_j_d, sizeof(int)*len);
				hipMalloc(&cuda_points, sizeof(Point) * (bd_size + 2) * (bd_size + 2));
				std::cout << "Cuda malloc done" << std::endl;



				hipMemcpy(c_i_d, c_i, sizeof(int)*len, hipMemcpyHostToDevice);
				hipMemcpy(c_j_d, c_j, sizeof(int)*len, hipMemcpyHostToDevice);
				hipMemcpy(cuda_points, points, sizeof(Point) * (bd_size + 2) * (bd_size + 2), hipMemcpyHostToDevice);

				CudaBoard* board = get_board(sequence, bd_size);
				board->print_board();

				std::cout << "ready to run cuda code run_simulation()" << std::endl;
				hipEventRecord(start);
				run_simulation <<< 1, 1, sizeof(Point)*(bd_size + 2)*(bd_size + 2) >>> (c_i_d, c_j_d, len, cuda_win_increase, cuda_points, bd_size, time(NULL));
				hipEventRecord(stop);
				printf("return : %s\n", hipGetErrorString(hipDeviceSynchronize()));

				memoryUsage();
				hipDeviceSynchronize();
				int* win_increase = new int[1];
				hipMemcpy(win_increase, cuda_win_increase, sizeof(int), hipMemcpyDeviceToHost);

				hipEventSynchronize(stop);
				float milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start, stop);

				printf("time: %f\n", milliseconds);
				printf("win: %d\n", *win_increase);
				hipDeviceReset();

				children[i]->wins += *win_increase;
				children[i]->sims += MAX_TRIAL_H;
				//printf("win:%d, sims:%d\n", children[i]->wins, children[i]->sims);
				back_propagation(children[i], *win_increase, MAX_TRIAL_H);
				delete [] win_increase;
				if (checkAbort())break;

			}
		}
		if (checkAbort()) break;
	}
	std::cout << "run_iteration end:" << std::endl;
	delete [] c_i;
	delete [] c_j;
}

bool Mcts::checkAbort() {
	if (!abort) {
		// abort = mcts_timer.GetTime() > maxTime;
	}
	return abort;
}

std::vector<Point> Mcts::generateAllMoves(CudaBoard* cur_board) {
	Point* point = createPoints(bd_size);
	std::vector<Point> moves_vec = cur_board->get_next_moves_host(point);
	int len = moves_vec.size();

	/* NOTE: point has not been freed yet !!!!!*/

	return moves_vec;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	Point* point = createPoints(bd_size);
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board((*it), point);
	}
	return bd;
}


__device__ void deleteAllMoves(Deque<Point*>* moves) {
	Deque<Point*>::iterator it = moves->begin();
	for (; it != moves->end(); it++) {
		Point* p = *it;
		delete p;
	}
}

void Mcts::deleteAllMoves(std::vector<Point*> moves) {
	for (std::vector<Point*>::iterator it = moves.begin(); it != moves.end(); it++) {
		delete *it;
	}
}

__device__ __host__ Point* createPoints(int bd_size) {
	int len = bd_size + 2;
	Point* point = static_cast<Point*> (malloc(sizeof(Point) * len * len));
	for (int i = 0; i < len; i++) {
		for (int j = 0; j < len; j++) {
			point[i * len + j] = Point(i, j);
		}
	}
	return point;
}

void memoryUsage() {
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;




	double free_db = (double)free_byte ;

	double total_db = (double)total_byte ;

	double used_db = total_db - free_db ;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

	       used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}

