#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "mcts.h"
#include "CudaGo.h" 
#include "deque.h"

//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

__constant__ int MAX_TRIAL_H = 4;
int MAX_TRIAL = 5;

static int grid_dim = 4;
static int block_dim = 1;
static int THREADS_NUM = grid_dim * block_dim;

bool checkAbort();
__device__ bool checkAbortCuda(bool* abort, clock_t startTime, double timeLeft);
__global__ void run_simulation(int* iarray, int* jarray, int len, int* win_increase, double timeLeft, int bd_size, unsigned int seed);
__device__ __host__ Point*** createPoints(int bd_size);
__device__ __host__ void deletePoints(Point*** point, int bd_size);
__device__ void deleteAllMoves(Deque<Point*>* moves);

void memoryUsage();

Point Mcts::run() {
	// mcts_timer.Start();
	int iter = 0;
	while (iter < 1) {
		run_iteration(root);
		if (checkAbort()) break;
		iter++;
	}
	double maxv = -1.0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}
	// std::cout << "Total simulation runs:" << totalSimu << std::endl;
	return best->get_sequence().back();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;

	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation
__global__ void run_simulation(int* iarray, int* jarray, int len, int* win_increase, double timeLeft, int bd_size, unsigned int seed) {
	// TODO: use shared memory for point
	clock_t very_start = 0;

	__shared__ Point*** globalPoints;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadIdx.x == 0){
		globalPoints = createPoints(bd_size);
	}
	if(index == 0)
	printf("time cp1:%lld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);
	__syncthreads();
	extern __shared__ int wins[]; //size declared win kernel is called.
	bool abort = false;
	win_increase[index] = 0;
	clock_t cudaStartTime = std::clock();
	int times = 0;
	for(int trial = 0; trial < MAX_TRIAL_H; trial++){
		CudaBoard* board = new CudaBoard(bd_size);
		if(index == 0)
		printf("time cp2:%ld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);

		for (int i = 0; i < len; i++) {
			board->update_board(globalPoints[iarray[i]][jarray[i]], globalPoints);
		}
		if(index == 0)
		printf("time cp3:%ld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);

		// for (int i = 0; i < 100; i++) {
		// 	Deque<Point*>* moves = board->get_next_moves_device(globalPoints);
		// 	board->update_board(moves->front(), globalPoints);
		// }

		COLOR player = board->ToPlay();
		hiprandState_t state;
		hiprand_init(seed + index, 0, 0, &state);

		// bool timeout = false;
		*win_increase = 0;
		int step = 0;
		clock_t ttime = std::clock();
		if(index == 0)
		printf("time cp4:%lld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);
		while (true) {
			Deque<Point*>* moves = board->get_next_moves_device(globalPoints);
			if (moves->size() == 0) {
				break;
			}
			
			Point* nxt_move = (*moves)[hiprand(&state) % moves->size()];
			//Point* nxt_move = moves->front();
			board->update_board(nxt_move, globalPoints);
			step++;
			if(checkAbortCuda(&abort, cudaStartTime, timeLeft))break; 
		}
		if(index == 0){
			printf("time cp5:%ld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);
		}
		

		// printf("id:%d, step:%d, timeleft:%lf\n", index, step, timeLeft);
		times++;
		// printf("time used for one game:%lf\n", 1000.0 * (std::clock() - ttime) / CLOCKS_PER_SEC);
		if(checkAbortCuda(&abort, cudaStartTime, timeLeft))break;
		int score = board->score(); // Komi set to 0
		if(index == 0)
		printf("time cp6:%ld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);
		if ((score > 0 && player == BLACK)
		        || (score < 0 && player == WHITE)) {
			win_increase[index]++;
		}

		delete board;
	}
	if(index == 0) 
		printf("num of trial done:%d\n",times);
	if(index == 0) 	printf("time cp7:%ld\n", (std::clock() - very_start) / CLOCKS_PER_SEC);
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(), bd_size);

	std::vector<Point*> moves_vec = generateAllMoves(cur_board);
	std::cout << "moves generated:" << moves_vec.size() << std::endl;
	while (moves_vec.size() > 0) {
		Point* nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), *nxt_move));
		moves_vec.pop_back();
		delete nxt_move;
	}
	std::cout << "children add done" << std::endl;
	deleteAllMoves(moves_vec);
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	int total = bd_size * bd_size;
	int* c_i = new int[total];
	int* c_j = new int[total];
	int* c_i_d; // device
	int* c_j_d; // device

	std::cout << "run_iteration start:" << std::endl;

	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			//	std::cout<<"select f:"<<f<<std::endl;
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);

			std::vector<TreeNode*> children = f->get_children();
			for (size_t i = 0; i < children.size(); i++) {
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);


				// hipMemset(cuda_win_increase, 0, sizeof(int)*THREADS_NUM);
				hipMalloc(&c_i_d, sizeof(int)*total);
				hipMalloc(&c_j_d, sizeof(int)*total);
				std::cout << "Cuda malloc done" << std::endl;

				std::vector<Point> sequence = children[i]->get_sequence();
				int len = sequence.size();
    			thrust::device_ptr<int> cuda_win_increase = thrust::device_malloc<int>(THREADS_NUM);

				for (int it = 0; it < len; it++) {
					c_i[it] = sequence[it].i;
					c_j[it] = sequence[it].j;
				}

				hipMemcpy(c_i_d, c_i, sizeof(int)*len, hipMemcpyHostToDevice);
				hipMemcpy(c_j_d, c_j, sizeof(int)*len, hipMemcpyHostToDevice);

				CudaBoard* board = get_board(sequence, bd_size);
				board->print_board();

				double timeLeft = maxTime - 1000.0*(std::clock() - startTime)/double(CLOCKS_PER_SEC);
				// printf("startTime before kernel:%ld\n",startTime);
				// printf("current clock time:%ld\n",clock());
				// printf("CLOCKS_PER_SEC:%ld\n",CLOCKS_PER_SEC);
				// printf("timeLeft before kernel:%lf\n", timeLeft);
				// std::cout << "ready to run cuda code run_simulation()" << std::endl;
				hipEventRecord(start);
				run_simulation <<<grid_dim, block_dim>>>(c_i_d, c_j_d, len, cuda_win_increase.get(), timeLeft, bd_size, time(NULL));
				hipEventRecord(stop);
				printf("return : %s\n", hipGetErrorString(hipDeviceSynchronize()));

				memoryUsage();
				hipDeviceSynchronize();

    			thrust::device_ptr<int> d_output = thrust::device_malloc<int>(THREADS_NUM);
    
    			thrust::exclusive_scan(cuda_win_increase, cuda_win_increase + THREADS_NUM, d_output);

    			hipDeviceSynchronize();
    			
				int* win_increase = new int[1];
    			hipMemcpy(win_increase, d_output.get()+THREADS_NUM-1, sizeof(int), hipMemcpyDeviceToHost);
			    thrust::device_free(d_output);

				hipEventSynchronize(stop);
				float milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start, stop);


				printf("time measured in CPU: %lf\n", milliseconds);
				// for(int w = 0 ; w < THREADS_NUM; w++){
				// 	printf("win[i]: %d\n", win_increase[w]);
				// }
				

				hipDeviceReset();
				children[i]->wins += win_increase[0];
				children[i]->sims += MAX_TRIAL*THREADS_NUM;
				printf("win:%d, sims:%d\n", children[i]->wins, children[i]->sims);
				back_propagation(children[i], win_increase[0], MAX_TRIAL);
				delete win_increase;
				// if (checkAbort())break;

			}
		}
		if (checkAbort()) break;
	}
	std::cout << "run_iteration end:" << std::endl;
}

__device__ bool checkAbortCuda(bool* abort, clock_t cudaStartTime, double timeLeft){
	if (!(*abort)) {
		*abort = 1000.0 * (std::clock() - cudaStartTime) / CLOCKS_PER_SEC > timeLeft;
	}

	if(*abort) printf("is aborted in device, timeLeft:%lf, startTime:%d\n",timeLeft, cudaStartTime);
	else{
		printf("not aborted yet. lhs:%lf\n", 1000.0 * (std::clock() - cudaStartTime) / CLOCKS_PER_SEC);
	}
	return *abort;
}

bool Mcts::checkAbort() {
	if (!abort) {
		abort = 1000.0 * (std::clock() - startTime) / CLOCKS_PER_SEC > maxTime;
	}
	if(abort) printf("is aborted in host\n");
	return abort;
}

std::vector<Point*> Mcts::generateAllMoves(CudaBoard* cur_board) {
	Point*** point = createPoints(bd_size);

	std::vector<Point*> moves_vec = cur_board->get_next_moves_host(point);
	int len = moves_vec.size();

	/* NOTE: point has not been freed yet !!!!!*/

	return moves_vec;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	Point*** point = createPoints(bd_size);
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board(&(*it), point);
	}
	deletePoints(point, bd_size);
	return bd;
}


__device__ void deleteAllMoves(Deque<Point*>* moves) {
	Deque<Point*>::iterator it = moves->begin();
	for (; it != moves->end(); it++) {
		Point* p = *it;
		delete p;
	}
}

void Mcts::deleteAllMoves(std::vector<Point*> moves) {
	for (std::vector<Point*>::iterator it = moves.begin(); it != moves.end(); it++) {
		delete *it;
	}
}

__device__ __host__ Point*** createPoints(int bd_size) {
	int len = bd_size + 2;
	Point*** point = static_cast<Point***> (malloc(sizeof(Point*) * len));
	for (int i = 0; i < len; i++) {
		point[i] = static_cast<Point**> (malloc(sizeof(Point*) * len));
		for (int j = 0; j < len; j++) {
			point[i][j] = (Point*)malloc(sizeof(Point));
			point[i][j]->i = i;
			point[i][j]->j = j;
		}
	}
	return point;
}

__device__ __host__ void deletePoints(Point*** point, int bd_size) {
	for (int i = 0; i < bd_size + 2; i++) {
		for (int j = 0; j < bd_size + 2; j++) {
			delete point[i][j];
		}
		free(point[i]);
	}
	free(point);
}

void memoryUsage() {
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;




	double free_db = (double)free_byte ;

	double total_db = (double)total_byte ;

	double used_db = total_db - free_db ;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

	       used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}

