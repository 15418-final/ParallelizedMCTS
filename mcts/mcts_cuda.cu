#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "mcts.h"
#include "CudaGo.h" 
#include "deque.h"
#include "point.h"

//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

__constant__ int MAX_TRIAL_H = 5;
int MAX_TRIAL = 5;

static int grid_dim = 8;
static int block_dim = 8;
static int THREADS_NUM = grid_dim * block_dim;


int wrapSequence(std::vector<TreeNode*> children, Point* &existingPath, Point* &allNxtMoves);
bool checkAbort();
__device__ bool checkAbortCuda(bool* abort, clock_t startTime, double timeLeft);
__global__ void run_simulation(Point* existingPath, int pathLen, Point* allNxtMoves, int len, int* win_increase, Point* parray, int bd_size, unsigned int seed);
__device__ __host__ Point* createPoints(int bd_size);

void memoryUsage();

Point Mcts::run() {
	// mcts_timer.Start();
	size_t heapszie = 128 * 1024 * 1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, heapszie);

	while (true) {
		run_iteration(root);
		if (checkAbort()) break;
	}
	double maxv = -1.0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}
	// std::cout << "Total simulation runs:" << totalSimu << std::endl;
	return best->get_sequence().back();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;

	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation

__global__ void run_simulation(Point* existingPath, int pathLen, Point* allNxtMoves, int len, int* win_increase, Point* parray, double timeLeft, int bd_size, unsigned int seed) {
	// TODO: use shared memory for point
	// __shared__ Point* point;
	
	// if (threadIdx.x == 0) {
	// 	memcpy(point, parray, sizeof(Point)*(bd_size+2)*(bd_size+2));
	// }
	// __syncthreads();

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// __shared__ Point* sequence;
	// if(blockIdx.x < len && threadIdx.x == 0){
	// 	printf("children size:%d\n", len);
	// 	printf("blockIdx:%d\n", blockIdx.x);
	// 	printf("in global: %d,%d\n", allNxtMoves[blockIdx.x].i, allNxtMoves[blockIdx.x].j);
		
	// 	printf("done\n");
	// }
	win_increase[blockIdx.x] = 0;
	// __syncthreads();

	clock_t cudaStartTime = clock();
	CudaBoard* initBoard = new CudaBoard(bd_size);
	COLOR player = initBoard->ToPlay();
	hiprandState_t state;
	hiprand_init(seed + index, 0, 0, &state);
	for (int i = 0; i < pathLen; i++) {
		initBoard->update_board(parray[existingPath[i].i*(bd_size+2) + existingPath[i].j], parray);
	}
	for(int i = 0; i < MAX_TRIAL_H; i++){
		CudaBoard board = *initBoard;
		bool abort = false;
		int times = 0;
		int step = 0;
		
		board.update_board(parray[allNxtMoves[blockIdx.x].i * (bd_size+2) + allNxtMoves[blockIdx.x].j], parray);
	
		while (true && step < 300) {
			Deque<Point>* moves = board.get_next_moves_device(parray);
			if (moves->size() == 0) {
				break;
			}
			Point nxt_move = (*moves)[hiprand(&state) % moves->size()];
			board.update_board(nxt_move, parray);
			step++;
			// if(checkAbortCuda(&abort, cudaStartTime, timeLeft))break;
		}
		times++;
			// printf("time used for one game:%lf\n", 1000.0 * (std::clock() - ttime) / CLOCKS_PER_SEC);
			// if(checkAbortCuda(&abort, cudaStartTime, timeLeft))break;
		int score = board.score(); // Komi set to 0

		if ((score > 0 && player == BLACK)
		        || (score < 0 && player == WHITE)) {
			atomicInc((unsigned int*)&(win_increase[blockIdx.x]), 10000000);
		}
		printf("win in block %d: %d\n", blockIdx.x, win_increase[blockIdx.x]);
	
	}
	delete initBoard;
	// if(index == 0) 
	// 	printf("num of trial done:%d\n",times);
	// if(index == 0) 	printf("time cp7:%ld\n", (std::clock()) / CLOCKS_PER_SEC);
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(), bd_size);

	std::vector<Point> moves_vec = generateAllMoves(cur_board);
	while (moves_vec.size() > 0) {
		Point nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), nxt_move));
		moves_vec.pop_back();
	}
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	int total = bd_size * bd_size;

	Point* points = createPoints(bd_size);

	std::cout << "run_iteration start:" << std::endl;

	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);
			int len = f->get_children().size();
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			Point* cuda_points = NULL;
			thrust::device_ptr<int> cuda_win_increase = thrust::device_malloc<int>(grid_dim);

			hipMalloc(&cuda_points, sizeof(Point) * (bd_size + 2) * (bd_size + 2));
			// std::cout << "Cuda malloc done" << std::endl;

			hipMemcpy(cuda_points, points, sizeof(Point) * (bd_size + 2) * (bd_size + 2), hipMemcpyHostToDevice);
			
			double timeLeft = maxTime - 1000.0*(std::clock() - startTime)/double(CLOCKS_PER_SEC);
				// printf("startTime before kernel:%ld\n",startTime);
				// printf("current clock time:%ld\n",clock());
				// printf("CLOCKS_PER_SEC:%ld\n",CLOCKS_PER_SEC);
				// printf("timeLeft before kernel:%lf\n", timeLeft);
				// std::cout << "ready to run cuda code run_simulation()" << std::endl;
			hipEventRecord(start);

			Point* existingPath = NULL;
			Point* allNxtMoves = NULL;
			int pathLen = wrapSequence(f->get_children(), existingPath, allNxtMoves);

			run_simulation <<< grid_dim, block_dim >>> (existingPath, pathLen, allNxtMoves, len, cuda_win_increase.get(), cuda_points, timeLeft, bd_size, time(NULL));

			hipEventRecord(stop);
			printf("return : %s\n", hipGetErrorString(hipDeviceSynchronize()));

			hipDeviceSynchronize();
			memoryUsage();

    			
			int* win_increase = new int[grid_dim];
    		hipMemcpy(win_increase, cuda_win_increase.get(), sizeof(int)*grid_dim, hipMemcpyDeviceToHost);


			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);

			printf("time measured in CPU: %lf\n", milliseconds);
			// for(int i = 0; i < grid_dim; i++){
			// 	printf("win[i]: %d\n", win_increase[i]);
			// }
			

			hipDeviceReset();
			for(int i = 0; i < grid_dim; i++){
				f->get_children()[i]->wins += win_increase[i];
				f->get_children()[i]->sims += MAX_TRIAL*grid_dim;
				back_propagation(f->get_children()[i], f->get_children()[i]->wins, f->get_children()[i]->sims);
			}

			
			delete [] win_increase;
			if (checkAbort())break;
		}
		if (checkAbort()) break;
	}
	std::cout << "run_iteration end:" << std::endl;
}

__device__ bool checkAbortCuda(bool* abort, clock_t cudaStartTime, double timeLeft){
	if (!(*abort)) {
		*abort = 1000.0 * (std::clock() - cudaStartTime) / CLOCKS_PER_SEC > timeLeft;
	}

	// if(*abort) printf("is aborted in device, timeLeft:%lf, startTime:%d\n",timeLeft, cudaStartTime);
	// else{
	// 	printf("not aborted yet. lhs:%lf\n", 1000.0 * (std::clock() - cudaStartTime) / CLOCKS_PER_SEC);
	// }
	return *abort;
}

bool Mcts::checkAbort() {
	if (!abort) {
		abort = 1000.0 * (std::clock() - startTime) / CLOCKS_PER_SEC > maxTime;
	}
	if(abort) printf("is aborted in host\n");
	return abort;
}

std::vector<Point> Mcts::generateAllMoves(CudaBoard* cur_board) {
	Point* point = createPoints(bd_size);
	std::vector<Point> moves_vec = cur_board->get_next_moves_host(point);
	int len = moves_vec.size();

	/* NOTE: point has not been freed yet !!!!!*/

	return moves_vec;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	Point* point = createPoints(bd_size);
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board((*it), point);
	}
	return bd;
}



void Mcts::deleteAllMoves(std::vector<Point*> moves) {
	for (std::vector<Point*>::iterator it = moves.begin(); it != moves.end(); it++) {
		delete *it;
	}
}

__device__ __host__ Point* createPoints(int bd_size) {
	int len = bd_size + 2;
	Point* point = static_cast<Point*> (malloc(sizeof(Point) * len * len));
	for (int i = 0; i < len; i++) {
		for (int j = 0; j < len; j++) {
			point[i * len + j] = Point(i, j);
		}
	}
	return point;
}

Deque<Point> vec2deq(std::vector<Point> vp){
	Deque<Point> rst;
	for(int i = 0; i < vp.size(); i++){
		rst.push_back(vp[i]);
	}
	return rst;
}

int wrapSequence(std::vector<TreeNode*> children, Point* &existingPath, Point* &allNxtMoves){
	hipMalloc((void**)&existingPath, children[0]->get_sequence().size()*sizeof(Point));
	hipMalloc((void**)&allNxtMoves, sizeof(Point)*children.size());

	int commonPathLen = children[0]->get_sequence().size()-1;
	Point* temp = (Point*)malloc(sizeof(Point)*commonPathLen);
	for(int i = 0; i < commonPathLen; i++){
		memcpy(temp+i, &(children[0]->get_sequence()[i]), sizeof(Point));
	}
	hipMemcpy(existingPath, temp, sizeof(Point)*commonPathLen, hipMemcpyHostToDevice);

	temp = (Point*)realloc(temp, sizeof(Point) * children.size());


	for(int i = 0; i < children.size(); i++){
		memcpy(temp+i, &(children[i]->get_sequence().back()), sizeof(Point));
	}
	hipMemcpy(allNxtMoves, temp, sizeof(Point)*children.size(), hipMemcpyHostToDevice);
	delete temp;
	return commonPathLen;
}

void memoryUsage() {
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;




	double free_db = (double)free_byte ;

	double total_db = (double)total_byte ;

	double used_db = total_db - free_db ;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

	       used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}

