#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/extrema.h>
#include <stdint.h>

#include "mcts.h"
#include "CudaGo.h"
#include "deque.h"
#include "point.h"

#define BILLION 1000000000L
#define MILLION 1000000.0
//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

__constant__ int MAX_TRIAL_H = 1;
#define MAX_STEP 300 // avoid repeat game
__constant__ double CLOCK_RATE = 1215500.0; // For tesla K40

int MAX_TRIAL = 1;
#define MAX_GAME_TIME_9_9 1000.0
double MAX_GAME_TIME_11_11 = 4000.0;

static int grid_dim = 2880;
static int block_dim = 1;
static int THREADS_NUM = grid_dim * block_dim;
#define CPU_THREADS_NUM 59

bool checkAbort();
__device__ bool checkAbortCuda(long long int elapse, double timeLeft);
__global__ void run_simulation(int incre, int total, int* iarray, int* jarray, int* len, double* win_increase,
                               int* step, double* sim, int bd_size, unsigned int seed, double time);
__device__ __host__ Point* createPoints(int bd_size);
void* run_simulation_thread(void *arg);
void get_sequence(TreeNode* node, int* len, int* iarray, int*jarray);

void memoryUsage();

Point Mcts::run() {
	size_t heapszie = 1024 * 1024 * 1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, heapszie);

	while (true) {
		run_iteration(root);
		if (checkAbort()) break;
	}
	double maxv = -1.0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}

	return best->get_sequence().back();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -1.0;
	TreeNode* maxn = NULL;
	int n = node->sims;

	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation

__global__ void run_simulation(int incre, int total, int* iarray, int* jarray, int* len, double* win_increase,
                               int* step, double* sim, int bd_size, unsigned int seed, double time) {
	long long int start_game = clock64();
	int index = blockIdx.x;
	win_increase[index] = 0.0;
	step[index] = 0;
	sim[index] = 0;
	bool abort = false;

	hiprandState_t state;
	hiprand_init(seed + index, 0, 0, &state);

	CudaBoard board(bd_size);

	int id = index / incre;
	if (id >= total) id = total - 1;
	int p = 0;
	for (int i = 0; i < id; i++) {
		p += len[i];
	}
	for (int i = p; i < p + len[id]; i++) {
		board.update_board(Point(iarray[i], jarray[i]));
	}
	COLOR player = board.ToPlay();

	while (step[index] < MAX_STEP) {
		Point move = board.get_next_moves_device(hiprand_uniform(&state));
		if (move.i < 0) {
			break;
		}
		board.update_board(move);
		if ((clock64() - start_game) / CLOCK_RATE > time) {
			abort = true;
			break;
		}
		if (index == 0) {
			printf("time elapse:%f\n", (clock64() - start_game) / CLOCK_RATE);
		}
		step[index]++;
	}

	int score = board.score();
	if ((score > 0 && player == BLACK)
	        || (score < 0 && player == WHITE)) {
		if (abort) {
			win_increase[index] += (double)step[index] / MAX_STEP;
		} else {
			win_increase[index]++;
		}
	}

	if (abort) {
		sim[index] += (double) step[index] / MAX_STEP;
	} else {
		sim[index]++;
	}

//	if (index == 0) {printf("time:%f, step: %d\n", (clock64() - start_game) / CLOCK_RATE, step[index]);}
}

void* run_simulation_thread(void *arg) {
	thread_arg* a = static_cast<thread_arg*> (arg);
	int len = a->len;
	double timeLeft = a->time;
	int cur_step = 0;
	a->sim = 0.0;
	a->win = 0.0;
	bool abort = false;
	COLOR player;
	clock_t start = clock();
	CudaBoard* board;
	srand (time(NULL));

	while (true) {
		board =  new CudaBoard(a->bd_size);
		for (int i = 0; i < len; i++) {
			board->update_board(a->seq[i]);
		}
		player = board->ToPlay();
		if ((1000.0 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) break;

		cur_step = 0;
		while (cur_step < MAX_STEP) {
			std::vector<Point> moves = board->get_next_moves_host();
			if (moves.size() == 0) {
				break;
			}
			board->update_board(moves[rand() % moves.size()]);
			if ((1000.0 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) {
				abort = true;
				break;
			}
			cur_step++;
		}

		int score = board->score();
		if ((score > 0 && player == BLACK)
		        || (score < 0 && player == WHITE)) {
			if (abort) {
				a->win += (double)cur_step / MAX_STEP;
			} else {
				a->win++;
			}
		}

		if (abort) {
			a->sim += (double)cur_step / MAX_STEP;
		} else {
			a->sim++;
		}
		if ((MAX_GAME_TIME_9_9 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) break;
		delete board;
	}
	return;
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::update(TreeNode* node, double* sim, double* win, int incre, int thread_num) {
	std::vector<TreeNode*> children = node->get_children();
	for (int i = 0; i < thread_num; i++) {
		int id = i / incre;
		if (id >= children.size()) id = children.size() - 1;
		back_propagation(children[id], win[id], sim[id]);
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(), bd_size);

	std::vector<Point> moves_vec = cur_board->get_next_moves_host();
	while (moves_vec.size() > 0) {
		Point nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), nxt_move));
		moves_vec.pop_back();
	}
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	int total = bd_size * bd_size;
	int* c_i = new int[total * total];
	int* c_j = new int[total * total];
	int* cpu_len = new int[total];
	double* win_increase = new double[THREADS_NUM];
	double* sim_increase = new double[THREADS_NUM];
	int* step_increase = new int[THREADS_NUM];
	int* c_i_d; // device
	int* c_j_d; // device
	int* cuda_len;


	hipEvent_t start_event, stop;
	hipEventCreate(&start_event);
	hipEventCreate(&stop);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("clock :%d\n", prop.clockRate);

	hipMalloc(&cuda_len, sizeof(int) * total);
	hipMalloc(&c_i_d, sizeof(int)* total * total);
	hipMalloc(&c_j_d, sizeof(int)* total * total);

	// pthread_t* tids = (pthread_t*)malloc(sizeof(pthread_t) * CPU_THREADS_NUM);
	// thread_arg* args = (thread_arg*)malloc(sizeof(thread_arg) * CPU_THREADS_NUM);
	// for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
	// 	args[ti].seq = (Point*)malloc(sizeof(Point) * 300);
	// }

	std::cout << "run_iteration start:" << std::endl;
	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);

			get_sequence(f, cpu_len, c_i, c_j);
			int csize = f->get_children().size();
			int incre = THREADS_NUM / csize;

			// double thread_sim = 0;
			// for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
			// 	args[ti].len = (children[i]->get_sequence()).size();
			// 	for (int pi = 0; pi < args[ti].len; pi++) {
			// 		args[ti].seq[pi] = (children[i]->get_sequence())[pi];
			// 	}
			// 	args[ti].time = MAX_GAME_TIME_9_9;
			// 	args[ti].bd_size = bd_size;
			// 	args[ti].tid = ti;
			// 	pthread_create(&tids[ti], NULL, run_simulation_thread, (void *)(&args[ti]));
			// }

			thrust::device_ptr<double> cuda_win_increase = thrust::device_malloc<double>(THREADS_NUM);
			thrust::device_ptr<double> cuda_sim = thrust::device_malloc<double>(THREADS_NUM);
			thrust::device_ptr<int> cuda_step = thrust::device_malloc<int>(THREADS_NUM);

			hipMemcpy(c_i_d, c_i, sizeof(int)*total * total, hipMemcpyHostToDevice);
			hipMemcpy(c_j_d, c_j, sizeof(int)*total * total, hipMemcpyHostToDevice);
			hipMemcpy(cuda_len, cpu_len, sizeof(int)*total, hipMemcpyHostToDevice);

			uint64_t diff;
			clock_gettime(CLOCK_REALTIME, &end);
			diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
			double timeLeft = maxTime - diff / MILLION;

			hipEventRecord(start_event);
			run_simulation <<< grid_dim, block_dim >>> (incre, csize, c_i_d, c_j_d, cuda_len, cuda_win_increase.get(), cuda_step.get(), cuda_sim.get(),
			        bd_size, time(NULL), std::min(MAX_GAME_TIME_9_9, timeLeft));
			hipEventRecord(stop);

			printf("return : %s\n", hipGetErrorString(hipDeviceSynchronize()));

			// for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
			// 	pthread_join(tids[ti], NULL);
			// 	thread_sim += args[ti].sim;
			// }

			// printf("thread done, sim: %d\n", thread_sim);

			//memoryUsage();
			printf("THREADS_NUM:%d\n", THREADS_NUM);

			hipMemcpy(win_increase, cuda_win_increase.get(), sizeof(double) * THREADS_NUM, hipMemcpyDeviceToHost);
			hipMemcpy(step_increase, cuda_step.get(), sizeof(int) * THREADS_NUM, hipMemcpyDeviceToHost);
			hipMemcpy(sim_increase, cuda_sim.get(), sizeof(double) * THREADS_NUM, hipMemcpyDeviceToHost);

			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start_event, stop);

			double total_sim = 0.0;
			double total_win = 0.0;
			int total_step = 0;
			for (int i = 0; i < THREADS_NUM; i++) {
				total_sim += sim_increase[i];
				total_win += win_increase[i];
				total_step += step_increase[i];
			}
			printf("time measured in CPU: %lf\n", milliseconds);
			printf("win: %f\n", total_win);
			printf("step: %d\n", total_step);
			printf("gpu sim: %f\n", total_sim);
			//printf("gpu sim: %f, totoal:%f\n", total_sim[0], total_sim[0] + thread_sim);

			update(f, win_increase, sim_increase, incre, THREADS_NUM);

			thrust::device_free(cuda_win_increase);
			thrust::device_free(cuda_step);
			thrust::device_free(cuda_sim);
			if (checkAbort())break;
		}

		if (checkAbort()) break;
	}
	std::cout << "run_iteration end:" << std::endl;
	delete [] c_i;
	delete [] c_j;
	delete [] cpu_len;
	delete [] win_increase;
	delete [] sim_increase;
	delete [] step_increase;
}

void get_sequence(TreeNode* node, int* len, int* iarray, int*jarray) {
	std::vector<TreeNode*> children = node->get_children();
	int p = 0;
	for (size_t i = 0; i < children.size(); i++) {
		std::vector<Point> sequence = children[i]->get_sequence();
		len[i] = sequence.size();
		for (int it = 0; it < len[i]; it++) {
			iarray[it + p] = sequence[it].i;
			jarray[it + p] = sequence[it].j;
		}
		p += len[i];
	}
}

bool Mcts::checkAbort() {
	if (!abort) {
		uint64_t diff;
		clock_gettime(CLOCK_REALTIME, &end);
		diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
		abort = diff / MILLION > maxTime;
	}
	if (abort) printf("is aborted in host\n");
	return abort;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board((*it));
	}
	return bd;
}

__device__ __host__ Point* createPoints(int bd_size) {
	int len = bd_size + 2;
	Point* point = static_cast<Point*> (malloc(sizeof(Point) * len * len));
	for (int i = 0; i < len; i++) {
		for (int j = 0; j < len; j++) {
			point[i * len + j] = Point(i, j);
		}
	}
	return point;
}

void memoryUsage() {
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;

	double free_db = (double)free_byte ;

	double total_db = (double)total_byte ;

	double used_db = total_db - free_db ;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

	       used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}

