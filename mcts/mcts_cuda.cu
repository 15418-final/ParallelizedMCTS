#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>
#include <time.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/extrema.h>
#include <stdint.h>

#include "mcts.h"
#include "CudaGo.h"
#include "deque.h"
#include "point.h"

#define BILLION 1000000000L
#define MILLION 1000000.0
//Exploration parameter
double C = 1.4;
double EPSILON = 10e-6;

__constant__ int MAX_TRIAL_H = 1;
#define MAX_STEP 300 // avoid repeat game
__constant__ double CLOCK_RATE = 1215500.0; // For tesla K40

int MAX_TRIAL = 1;
#define MAX_GAME_TIME_9_9 1000.0
double MAX_GAME_TIME_11_11 = 4000.0;

static int grid_dim = 1;
static int block_dim = 1;
static int THREADS_NUM = grid_dim * block_dim;
static int CPU_THREADS_NUM = 59;

bool checkAbort();
__device__ bool checkAbortCuda(long long int elapse, double timeLeft);
__global__ void run_simulation(int* iarray, int* jarray, int len, double* win_increase, int* step, double* sim, int bd_size, unsigned int seed, double time);
__device__ __host__ Point* createPoints(int bd_size);
void* run_simulation_thread(void *arg);

void memoryUsage();

Point Mcts::run() {
	// mcts_timer.Start();
	size_t heapszie = 256 * 1024 * 1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, heapszie);

	while (true) {
		run_iteration(root);
		if (checkAbort()) break;
	}
	double maxv = -1.0;
	TreeNode* best = NULL;
	std::vector<TreeNode*> children = root->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON);
		if (v > maxv) {
			maxv = v;
			best = c;
		}
	}

	return best->get_sequence().back();
}

TreeNode* Mcts::selection(TreeNode* node) {
	std::cout << "selection begin" << std::endl;
	double maxv = -10000000;
	TreeNode* maxn = NULL;
	int n = node->sims;

	std::vector<TreeNode*> children = node->get_children();
	for (std::vector<TreeNode*>::iterator it = children.begin(); it != children.end(); it++) {
		TreeNode* c = *it;
		double v = (double)c->wins / (c->sims + EPSILON) + C * sqrt(log(n + EPSILON) / (c->sims + EPSILON));
		if (v > maxv) {
			maxv = v;
			maxn = c;
		}
	}
	std::cout << "selection end" << std::endl;
	return maxn;
}

// Typical Monte Carlo Simulation

__global__ void run_simulation(int* iarray, int* jarray, int len, double* win_increase, int* step, double* sim, int bd_size, unsigned int seed, double time) {
	long long int start_game = clock64();
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	win_increase[index] = 0.0;
	step[index] = 0;
	sim[index] = 0;
	bool abort = false;

	hiprandState_t state;
	hiprand_init(seed + index, 0, 0, &state);
	CudaBoard board(bd_size);
	for (int i = 0; i < len; i++) {
		board.update_board(Point(iarray[i], jarray[i]));
	}
	COLOR player = board.ToPlay();

	while (step[index] < MAX_STEP) {
		Point move = board.get_next_moves_device(hiprand_uniform(&state));
		if (move.i < 0) {
			break;
		}
		board.update_board(move);
		if ((clock64() - start_game) / CLOCK_RATE > time) {
			abort = true;
			break;
		}
		step[index]++;
	}

	int score = board.score();
	if ((score > 0 && player == BLACK)
	        || (score < 0 && player == WHITE)) {
		if (abort) {
			win_increase[index] += (double)step[index] / MAX_STEP;
		} else {
			win_increase[index]++;
		}
	}

	if (abort) {
		sim[index] += (double) step[index] / MAX_STEP;
	} else {
		sim[index]++;
	}
	if (index == 0) {printf("time:%f\n", (clock64() - start_game) / CLOCK_RATE);}
}

void* run_simulation_thread(void *arg) {
	thread_arg* a = static_cast<thread_arg*> (arg);
	int len = a->len;
	double timeLeft = a->time;
	int cur_step = 0;
	a->sim = 0.0;
	a->win = 0.0;
	bool abort = false;
	COLOR player;
	clock_t start = clock();
	CudaBoard* board;
	srand (time(NULL));

	while (true) {
		board =  new CudaBoard(a->bd_size);
		for (int i = 0; i < len; i++) {
			board->update_board(a->seq[i]);
		}
		player = board->ToPlay();
		if ((1000.0 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) break;

		cur_step = 0;
		while (cur_step < MAX_STEP) {
			std::vector<Point> moves = board->get_next_moves_host();
			if (moves.size() == 0) {
				break;
			}
			board->update_board(moves[rand() % moves.size()]);
			if ((1000.0 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) {
				abort = true;
				break;
			}
			cur_step++;
		}

		int score = board->score();
		if ((score > 0 && player == BLACK)
		        || (score < 0 && player == WHITE)) {
			if (abort) {
				a->win += (double)cur_step / MAX_STEP;
			} else {
				a->win++;
			}
		}

		if (abort) {
			a->sim += (double)cur_step / MAX_STEP;
		} else {
			a->sim++;
		}
		if ((MAX_GAME_TIME_9_9 * (clock() - start) / CLOCKS_PER_SEC) > timeLeft) break;
		delete board;
	}
	return;
}

void Mcts::back_propagation(TreeNode* node, int win_increase, int sim_increase) {
	bool lv = false;
	while (node->parent != NULL) {
		node = node->parent;
		node->sims += sim_increase;
		if (lv)node->wins += win_increase;
		lv = !lv;
	}
}

void Mcts::expand(TreeNode* node) {
	std::cout << "expand begin" << std::endl;
	CudaBoard* cur_board = get_board(node->get_sequence(), bd_size);

	std::vector<Point> moves_vec = cur_board->get_next_moves_host();
	while (moves_vec.size() > 0) {
		Point nxt_move = moves_vec.back();
		node->add_children(new TreeNode(node->get_sequence(), nxt_move));
		moves_vec.pop_back();
	}
	delete cur_board;

	std::cout << "expand end with children num:" << node->get_children().size() << std::endl;
}

void Mcts::run_iteration(TreeNode* node) {
	std::stack<TreeNode*> S;
	S.push(node);

	int total = bd_size * bd_size;
	int* c_i = new int[total];
	int* c_j = new int[total];
	double* win_increase = new double[1];
	double* total_sim = new double[1];
	int* total_step = new int[1];
	int* c_i_d; // device
	int* c_j_d; // device

	hipEvent_t start_event, stop;
	hipEventCreate(&start_event);
	hipEventCreate(&stop);

	hipMalloc(&c_i_d, sizeof(int)*total);
	hipMalloc(&c_j_d, sizeof(int)*total);

	pthread_t* tids = (pthread_t*)malloc(sizeof(pthread_t) * CPU_THREADS_NUM);
	thread_arg* args = (thread_arg*)malloc(sizeof(thread_arg) * CPU_THREADS_NUM);
	for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
		args[ti].seq = (Point*)malloc(sizeof(Point) * 300);
	}

	std::cout << "run_iteration start:" << std::endl;
	while (!S.empty()) {
		TreeNode* f = S.top();
		S.pop();
		if (!f->is_expandable()) {
			S.push(selection(f));
		} else {
			// expand current node, run expansion and simulation
			f->set_expandable(false);
			expand(f);

			std::vector<TreeNode*> children = f->get_children();
			for (size_t i = 0; i < children.size(); i++) {

				double thread_sim = 0;
				for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
					args[ti].len = (children[i]->get_sequence()).size();
					for (int pi = 0; pi < args[ti].len; pi++) {
						args[ti].seq[pi] = (children[i]->get_sequence())[pi];
					}
					args[ti].time = MAX_GAME_TIME_9_9;
					args[ti].bd_size = bd_size;
					args[ti].tid = ti;
					pthread_create(&tids[ti], NULL, run_simulation_thread, (void *)(&args[ti]));
				}

				std::vector<Point> sequence = children[i]->get_sequence();
				int len = sequence.size();

				for (int it = 0; it < len; it++) {
					c_i[it] = sequence[it].i;
					c_j[it] = sequence[it].j;
				}

				thrust::device_ptr<double> cuda_win_increase = thrust::device_malloc<double>(THREADS_NUM);
				thrust::device_ptr<double> cuda_sim = thrust::device_malloc<double>(THREADS_NUM);
				thrust::device_ptr<int> cuda_step = thrust::device_malloc<int>(THREADS_NUM);


				hipMemcpy(c_i_d, c_i, sizeof(int)*len, hipMemcpyHostToDevice);
				hipMemcpy(c_j_d, c_j, sizeof(int)*len, hipMemcpyHostToDevice);

				CudaBoard* board = get_board(sequence, bd_size);
				board->print_board();

				uint64_t diff;
				clock_gettime(CLOCK_REALTIME, &end);
				diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
				double timeLeft = maxTime - diff/MILLION;

				hipEventRecord(start_event);
				run_simulation <<< grid_dim, block_dim >>> (c_i_d, c_j_d, len, cuda_win_increase.get(), cuda_step.get(), cuda_sim.get(),
				        bd_size, time(NULL), std::min(MAX_GAME_TIME_9_9, timeLeft));
				hipEventRecord(stop);

				printf("return : %s\n", hipGetErrorString(hipDeviceSynchronize()));

				for (int ti = 0; ti < CPU_THREADS_NUM; ti++) {
					pthread_join(tids[ti], NULL);
					thread_sim += args[ti].sim;
				}

				printf("thread done, sim: %d\n", thread_sim);

				//memoryUsage();
				printf("THREADS_NUM:%d\n", THREADS_NUM);

				thrust::inclusive_scan(cuda_win_increase, cuda_win_increase + THREADS_NUM, cuda_win_increase);
				thrust::inclusive_scan(cuda_step, cuda_step + THREADS_NUM, cuda_step);
				thrust::inclusive_scan(cuda_sim, cuda_sim + THREADS_NUM, cuda_sim);

				hipMemcpy(win_increase, cuda_win_increase.get() + THREADS_NUM - 1, sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(total_step, cuda_step.get() + THREADS_NUM - 1, sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpy(total_sim, cuda_sim.get() + THREADS_NUM - 1, sizeof(double), hipMemcpyDeviceToHost);

				hipEventSynchronize(stop);
				float milliseconds = 0;
				hipEventElapsedTime(&milliseconds, start_event, stop);

				printf("time measured in CPU: %lf\n", milliseconds);
				printf("win: %f\n", win_increase[0]);
				printf("step: %d\n", total_step[0]);
				printf("gpu sim: %f, totoal:%f\n", total_sim[0], total_sim[0] + thread_sim);

				children[i]->wins += win_increase[0];
				children[i]->sims += MAX_TRIAL * THREADS_NUM;

				back_propagation(children[i], children[i]->wins, children[i]->sims);
				thrust::device_free(cuda_win_increase);
				thrust::device_free(cuda_step);
				thrust::device_free(cuda_sim);
				if (checkAbort())break;
			}
		}
		if (checkAbort()) break;
	}
	std::cout << "run_iteration end:" << std::endl;
	delete [] c_i;
	delete [] c_j;
}

bool Mcts::checkAbort() {
	if (!abort) {
		uint64_t diff;
		clock_gettime(CLOCK_REALTIME, &end);
		diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
		abort = diff / MILLION > maxTime;
	}
	if (abort) printf("is aborted in host\n");
	return abort;
}

CudaBoard* Mcts::get_board(std::vector<Point> sequence, int bd_size) {
	CudaBoard* bd = new CudaBoard(bd_size);
	for (std::vector<Point>::iterator it = sequence.begin(); it != sequence.end(); it++) {
		bd->update_board((*it));
	}
	return bd;
}

__device__ __host__ Point* createPoints(int bd_size) {
	int len = bd_size + 2;
	Point* point = static_cast<Point*> (malloc(sizeof(Point) * len * len));
	for (int i = 0; i < len; i++) {
		for (int j = 0; j < len; j++) {
			point[i * len + j] = Point(i, j);
		}
	}
	return point;
}

void memoryUsage() {
	size_t free_byte ;

	size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;

	double free_db = (double)free_byte ;

	double total_db = (double)total_byte ;

	double used_db = total_db - free_db ;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

	       used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}

